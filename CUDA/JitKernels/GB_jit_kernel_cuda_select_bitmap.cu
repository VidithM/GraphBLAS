#include "hip/hip_runtime.h"
using namespace cooperative_groups ;

#define tile_sz 32
#define log2_tile_sz 5

#include "GB_cuda_atomics.cuh"


__device__ __inline__ uint64_t GB_cuda_warp_sum_uint64
(
    thread_block_tile<tile_sz> tile,
    uint64_t value
)
{

    //--------------------------------------------------------------------------
    // sum value on all threads to a single value
    //--------------------------------------------------------------------------

    #if (tile_sz == 32)
    {
        // this is the typical case
        value += tile.shfl_down (value, 16) ;
        value += tile.shfl_down (value,  8) ;
        value += tile.shfl_down (value,  4) ;
        value += tile.shfl_down (value,  2) ;
        value += tile.shfl_down (value,  1) ;
    }
    #else
    {
        #pragma unroll
        for (int offset = tile_sz >> 1 ; offset > 0 ; offset >>= 1)
        {
            value += tile.shfl_down (value, offset) ;
        }
    }
    #endif

    return (value) ;
}

__inline__ __device__ uint64_t GB_block_Reduce
(
    thread_block g,
    uint64_t val
)
{
    static __shared__ uint64_t shared [tile_sz] ;
    int lane = threadIdx.x & (tile_sz-1) ;
    int wid  = threadIdx.x >> log2_tile_sz ;
    thread_block_tile<tile_sz> tile = tiled_partition<tile_sz>( g ) ;

    // Each warp performs partial reduction
    val = GB_cuda_warp_sum_uint64 (tile, val) ;

    // Wait for all partial reductions
    if (lane == 0)
    {
        shared [wid] = val ; // Write reduced value to shared memory
    }
    this_thread_block().sync() ;        // Wait for all partial reductions

    // for this to work, we need blockDim.x <= 32 * 32 ?
    val = (threadIdx.x < (blockDim.x >> LOG2_WARPSIZE)) ?  shared [lane] : 0 ;

    // Final reduce within first warp
    // for this to work, we need blockDim.x <= 32 * 32 ?
    val = GB_cuda_warp_sum_uint64 (tile, val) ;
    return (val) ;
}

__global__ void GB_cuda_select_bitmap_kernel
(
    int8_t *Cb_out,
    uint64_t *cnvals_out,
    GrB_Matrix A,
    const GB_void *thunk
)
{
    #if ( GB_DEPENDS_ON_X )
    const GB_A_TYPE *__restrict__ Ax = (GB_A_TYPE *) A->x ;
    #endif

    #if ( GB_A_IS_BITMAP )
    const int8_t *__restrict__ Ab = A->b ;
    #endif

    GB_A_NHELD (anz) ;
    int64_t nrows = A->vlen ;

    uint64_t my_keep = 0 ;
    int tid = blockIdx.x * blockDim.x + threadIdx.x ;
    int nthreads = blockDim.x * gridDim.x ;
    for (int64_t p = tid ; p < anz ; p += nthreads)
    {
        Cb_out [p] = 0 ;
        if (!GBB_A (Ab, p)) { continue; }

        #if ( GB_DEPENDS_ON_I )
        int64_t i = (p % nrows) ;
        #endif

        #if ( GB_DEPENDS_ON_J )
        int64_t j = (p / nrows) ;
        #endif
        
        #if ( GB_DEPENDS_ON_Y )
        GB_Y_TYPE y = * ((GB_Y_TYPE *) thunk) ;
        #endif

        GB_TEST_VALUE_OF_ENTRY (keep, p) ;
        if (keep) 
        {
            my_keep++ ;
            Cb_out [p] = 1 ;    
        } 
    }
    
    // can make this a warp-level synchronization?
    this_thread_block().sync() ;
    // compute cnvals for this block
    // IMPORTANT: every thread in the threadblock must participate in the warp reduction
    // for thread 0 to obtain the right result
    uint64_t block_keep = GB_block_Reduce (this_thread_block(), my_keep) ;
    // this can also be a warp-level synchronization?
    // (we only care about the result in warp 0, since that is where thread 0 is)
    this_thread_block().sync() ;

    if (threadIdx.x == 0)
    {
        // thread 0 updates global cnvals with atomics
        GB_cuda_atomic_add (cnvals_out, block_keep) ;
    }
}


extern "C"
{
    GB_JIT_CUDA_KERNEL_SELECT_BITMAP_PROTO (GB_jit_kernel) ;
}

GB_JIT_CUDA_KERNEL_SELECT_BITMAP_PROTO (GB_jit_kernel)
{
    dim3 grid (gridsz) ;
    dim3 block (blocksz) ;
    GB_cuda_select_bitmap_kernel <<<grid, block, 0, stream>>> (Cb, cnvals, A, ythunk) ;
    return (GrB_SUCCESS) ;
}